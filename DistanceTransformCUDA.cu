#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include <string.h>
#include <iostream>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime_api.h>

using namespace std;

typedef unsigned char uchar;
typedef unsigned short ushort;
typedef unsigned int uint;

#define eee(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

template<typename T>
__global__ void edt_cols(T* d_input, T* d_output, uint width, uint height)
{
	// x in range [0, width-1]	
	uint x = blockIdx.x * blockDim.x + threadIdx.x; 

	if(x >= width)
		return;
	
	extern __shared__ T g[];

	// Initialize val to either 0 or 'infinity'
	T val = (1-d_input[x]) * (width+height);
	g[0] = val;

	// Scan 1
	for (uint y = 1; y < height; y++)
	{
		val = (1 - d_input[y * width + x]) * (1 + val);
		g[y] = val;
	}

	// Scan 2	
	// y < height is the same as y >= 0, as this uint underflows
	for (uint y = height - 2; y < height; y--)
	{
		if (g[y] > val)
		{
			g[y] = 1 + val;
		}

		val = g[y];
	}
	
	for(uint y = 0; y < height; y++)
		d_output[y * width + x] = g[y];
}

template<typename T>
__global__ void edt_rows(T* d_output, uint width, uint height)
{
	uint x = blockIdx.x * blockDim.x + threadIdx.x;  // range [0, width-1]
	uint y = blockIdx.y * blockDim.y + threadIdx.y;  // range [0, height-1]

	if(x >= width)
		return;

	extern __shared__ T d_localG[];

	for (uint i = threadIdx.x; i < width; i += blockDim.x)
		d_localG[i] = d_output[y * width + i]; 

	__syncthreads();	

	T minDist = FLT_MAX;
	for (uint i = 0; i < width; i++)
	{
		minDist = fminf(minDist, (x-i)*(x-i) + d_localG[i] * d_localG[i]);
	}

	d_output[y * width + x] = sqrtf(minDist); 
}

template <typename T>
void runCUDA(T* h_inData, T* h_outData, uint width, uint height)
{
	size_t numBytes = height * width * sizeof(T);
	assert(numBytes > 0);

	T* d_inData;
	eee(hipMalloc((void **) &d_inData, numBytes));
	eee(hipMemcpy(d_inData, h_inData, numBytes, hipMemcpyHostToDevice));	

	T* d_outData;
	eee(hipMalloc((void **)&d_outData, numBytes));
	
	// TODO: Assert width/height are not too large to have a shared memory copy (due to SMEM size limit)

	dim3 colsGrid(width, 1, 1);
	dim3 colsThreads(1, 1, 1);

	uint threadsPerBlock = 1024;		
	dim3 rowsGrid(ceil((1.0f*width) / threadsPerBlock), height, 1);
	dim3 rowsThreads(min(width, 1024), 1, 1);

	uint numtrials = 10000;

	// Warmup
	for (int i = 0; i < (numtrials / 10); i++)
	{
		edt_cols<<<colsGrid, colsThreads, height * sizeof(T)>>>(d_inData, d_outData, width, height);
		edt_rows<<<rowsGrid, rowsThreads, width * sizeof(T)>>>(d_outData, width, height);
		eee(hipDeviceSynchronize());
	}

	auto start = std::chrono::high_resolution_clock::now();
	{
		for (int i = 0; i < numtrials; i++)
		{
			edt_cols<<<colsGrid, colsThreads, height * sizeof(T)>>>(d_inData, d_outData, width, height);
			edt_rows<<<rowsGrid, rowsThreads, width * sizeof(T)>>>(d_outData, width, height);
			eee(hipDeviceSynchronize());
		}
	}
	auto duration = std::chrono::high_resolution_clock::now() - start;
	long long ms = std::chrono::duration_cast<std::chrono::microseconds>(duration).count();
	printf("runCUDA executed in %lld microseconds\n", ms / numtrials);    
    
	eee(hipGetLastError());
	eee(hipMemcpy(h_outData, d_outData, numBytes, hipMemcpyDeviceToHost)); 
	
	eee(hipFree(d_inData));
	eee(hipFree(d_outData));	

	eee(hipProfilerStop());
	eee(hipDeviceReset());
}

int main(int argc, char **argv)
{
	printf("Starting\n");
	
	uint width = 256;
	uint height = 256;

	vector<float> inputData(width * height);
	vector<float> outputData(width * height);
	for (uint x = 0; x < width; x++)
	{
		for (uint y = 0; y < height; y++)
		{
			inputData[y * width + x] = (float)(x > 100 && x < 150 && y > 100 && y < 150 ? 1.0f : 0.0f); 
			inputData[y * width + x] = (float)(inputData[y * width + x] || abs((float)(x - y)) < 3 ? 1.0f : 0.0f);
		}
	}

	runCUDA(inputData.data(), outputData.data(), width, height);

	ofstream fout("input.dat", ios::out | ios::binary);
	fout.write((char*)inputData.data(), inputData.size() * sizeof(inputData[0]));
	fout.close();

	fout = ofstream("output.dat", ios::out | ios::binary);
	fout.write((char*)outputData.data(), outputData.size() * sizeof(outputData[0]));
	fout.close();
}